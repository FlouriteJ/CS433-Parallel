#include "hip/hip_runtime.h"
//
// include files
//

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime_api.h>


//
// kernel routine
// 

__global__ void my_first_kernel(float *x,float *y)
{
  int tid = threadIdx.x + blockDim.x*blockIdx.x;

  y[tid] = x[tid] + y[tid];
  
}


//
// main code
//

int main(int argc, const char **argv)
{
  float *h_x, *h_y, *d_x, *d_y;
  int   nblocks, nthreads, nsize, n; 

  // initialise card

  findCudaDevice(argc, argv);

  // set number of blocks, and threads per block

  nblocks  = 2;
  nthreads = 8;
  nsize    = nblocks*nthreads ;

  // allocate memory for array

  h_x = (float *)malloc(nsize*sizeof(float));
  for (n=0; n<nsize; n++) h_x[n] = n;
  h_y = (float *)malloc(nsize*sizeof(float));
  for (n=0; n<nsize; n++) h_y[n] = n;
  checkCudaErrors(hipMalloc((void **)&d_x, nsize*sizeof(float)));
  checkCudaErrors(hipMalloc((void **)&d_y, nsize*sizeof(float)));
  checkCudaErrors(hipMemcpy(d_x,h_x,nsize*sizeof(float),hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y,h_y,nsize*sizeof(float),hipMemcpyHostToDevice));
  for (n=0; n<nsize; n++) printf(" n,  x , y =  %d  %f  %f \n",n,h_x[n],h_y[n]);
  // execute kernel
  
  my_first_kernel<<<nblocks,nthreads>>>(d_x,d_y);
  getLastCudaError("my_first_kernel execution failed\n");

  // copy back results and print them out

  checkCudaErrors( hipMemcpy(h_y,d_y,nsize*sizeof(float),
                 hipMemcpyDeviceToHost) );

  for (n=0; n<nsize; n++) printf(" n,  result  =  %d  %f \n",n,h_y[n]);

  // free memory 

  checkCudaErrors(hipFree(d_x));
  checkCudaErrors(hipFree(d_y));
  free(h_x);
  free(h_y);

  // CUDA exit -- needed to flush printf write buffer

  hipDeviceReset();

  return 0;
}
