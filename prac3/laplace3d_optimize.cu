#include "hip/hip_runtime.h"
//
// Program to solve Laplace equation on a regular 3D grid
//

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime_api.h>

////////////////////////////////////////////////////////////////////////
// define kernel block size
////////////////////////////////////////////////////////////////////////

int BLOCK_X, BLOCK_Y;
////////////////////////////////////////////////////////////////////////
// include kernel function
////////////////////////////////////////////////////////////////////////

#include <laplace3d_kernel_optimize.h>


int main(int argc, const char **argv){

  // 'h_' prefix - CPU (host) memory space

  int    NX=256, NY=256, NZ=256, REPEAT=50,
         bx, by, i, j, k, ind;
  float  *h_u1, *h_u2, *h_u3, *h_foo, err;

  // 'd_' prefix - GPU (device) memory space

  float  *d_u1, *d_u2, *d_foo;

  printf("\nGrid dimensions: %d x %d x %d\n", NX, NY, NZ);

  // initialise card

  findCudaDevice(argc, argv);

  // initialise CUDA timing

  float milli;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  
  for (BLOCK_X = 1;BLOCK_X<=256;BLOCK_X*=2){
	  for (BLOCK_Y = 1;BLOCK_Y<=256;BLOCK_Y*=2){
		  // allocate memory for arrays
		  
		  if (BLOCK_X * BLOCK_Y > 1024) break;

		  h_u1 = (float *)malloc(sizeof(float)*NX*NY*NZ);
		  h_u2 = (float *)malloc(sizeof(float)*NX*NY*NZ);
		  h_u3 = (float *)malloc(sizeof(float)*NX*NY*NZ);
		  checkCudaErrors( hipMalloc((void **)&d_u1, sizeof(float)*NX*NY*NZ) );
		  checkCudaErrors( hipMalloc((void **)&d_u2, sizeof(float)*NX*NY*NZ) );

		  // initialise u1

		  for (k=0; k<NZ; k++) {
			for (j=0; j<NY; j++) {
			  for (i=0; i<NX; i++) {
				ind = i + j*NX + k*NX*NY;

				if (i==0 || i==NX-1 || j==0 || j==NY-1|| k==0 || k==NZ-1)
				  h_u1[ind] = 1.0f;           // Dirichlet b.c.'s
				else
				  h_u1[ind] = 0.0f;
			  }
			}
		  }

		  // copy u1 to device

		  checkCudaErrors( hipMemcpy(d_u1, h_u1, sizeof(float)*NX*NY*NZ,
									  hipMemcpyHostToDevice));

		  // Set up the execution configuration

		  bx = ceil((NX-1.0)/BLOCK_X);
		  by = ceil((NY-1.0)/BLOCK_Y);

		  dim3 dimGrid(bx,by);
		  dim3 dimBlock(BLOCK_X,BLOCK_Y);

		  // printf("\n dimGrid  = %d %d %d \n",dimGrid.x,dimGrid.y,dimGrid.z);
		  // printf(" dimBlock = %d %d %d \n",dimBlock.x,dimBlock.y,dimBlock.z);

		  // Execute GPU kernel

		  hipEventRecord(start);

		  for (i = 1; i <= REPEAT; ++i) {
			GPU_laplace3d<<<dimGrid, dimBlock>>>(NX, NY, NZ, d_u1, d_u2, BLOCK_X, BLOCK_Y);
			getLastCudaError("GPU_laplace3d execution failed\n");

			d_foo = d_u1; d_u1 = d_u2; d_u2 = d_foo;   // swap d_u1 and d_u2
		  }

		  hipEventRecord(stop);
		  hipEventSynchronize(stop);
		  hipEventElapsedTime(&milli, start, stop);
		  printf("%d %d  %.2f \n", BLOCK_X, BLOCK_Y, milli);

		  // Read back GPU results

		  checkCudaErrors( hipMemcpy(h_u2, d_u1, sizeof(float)*NX*NY*NZ,
									  hipMemcpyDeviceToHost) );

		 // Release GPU and CPU memory

		  checkCudaErrors( hipFree(d_u1) );
		  checkCudaErrors( hipFree(d_u2) );
		  free(h_u1);
		  free(h_u2);
		  free(h_u3);		  
	  }
  }


  hipDeviceReset();
}
